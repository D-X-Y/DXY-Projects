#include "hip/hip_runtime.h"
//copied from https://github.com/gidariss/caffe_LocNet/blob/d2ba49552068958556b98ba382610ea865add17c/src/caffe/layers/region_pooling_layer.cu

#include "luaT.h"
#include "THC.h"

#include <lua.h>
#include "THCGeneral.h"

#define CAFFE_CUDA_NUM_THREADS 1024

// CUDA: various checks for different function calls.
#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
	if(error != hipSuccess) { printf("CUDA ERROR. %s\n", hipGetErrorString(error)); }; \
  } while (0)

// CUDA: number of blocks for threads.
inline int CAFFE_GET_BLOCKS(const int N) {
  return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

// CUDA: check for error after kernel execution and exit loudly if there is one.
#define CUDA_POST_KERNEL_CHECK CUDA_CHECK(hipPeekAtLastError())

template <typename Dtype>
__global__ void ROIPoolForward(const int nthreads, const Dtype* bottom_data,
    const Dtype spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois, Dtype* top_data, int* argmax_data) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		// (n, c, ph, pw) is an element in the pooled output
		int pw = index % pooled_width;
		int ph = (index / pooled_width) % pooled_height;
		int c = (index / pooled_width / pooled_height) % channels;
		int n = index / pooled_width / pooled_height / channels;

		// For each ROI R = [batch_index, x_outer_1, y_outer_1, x_outer_2, y_outer_2, x_inner_1, y_inner_1, x_inner_2, y_inner_2]: 
		// where R_outer = [x_outer_1, y_outer_1, x_outer_2, y_outer_2] is the outer rectangle of the region and 
		// R_inner = [x_inner_1, y_inner_1, x_inner_2, y_inner_2] is the inner rectangle of the region
		// max pooler over R by ignoring (setting to zero) the activations that lay inside the inner rectangle R_inner

		bottom_rois += n * 9;
		int roi_batch_ind = bottom_rois[0];


		// outer rectangle of the region
		int roi_start_w   = int(bottom_rois[1] );//* spatial_scale);
		int roi_start_h   = int(bottom_rois[2] );//* spatial_scale);
		int roi_end_w     = int(bottom_rois[3] );//* spatial_scale);
		int roi_end_h     = int(bottom_rois[4] );//* spatial_scale);

		// inner rectangle of the region
		int roi_start_w_in = int(bottom_rois[5]);//* spatial_scale);
		int roi_start_h_in = int(bottom_rois[6]);//* spatial_scale);
		int roi_end_w_in   = int(bottom_rois[7]);//* spatial_scale);
		int roi_end_h_in   = int(bottom_rois[8]);//* spatial_scale);

		// Force malformed ROIs to be 1x1
		int roi_width  = max(roi_end_w - roi_start_w + 1, 1);
		int roi_height = max(roi_end_h - roi_start_h + 1, 1);
		Dtype bin_size_h = static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
		Dtype bin_size_w = static_cast<Dtype>(roi_width)  / static_cast<Dtype>(pooled_width);

		const int hstart = min(height, max(0, static_cast<int>(floor(static_cast<Dtype>(ph)   * bin_size_h)) + roi_start_h));
		const int hend   = min(height, max(0, static_cast<int>(ceil( static_cast<Dtype>(ph+1) * bin_size_h)) + roi_start_h));
		const int wstart = min(width,  max(0, static_cast<int>(floor(static_cast<Dtype>(pw)   * bin_size_w)) + roi_start_w));
		const int wend   = min(width,  max(0, static_cast<int>(ceil( static_cast<Dtype>(pw+1) * bin_size_w)) + roi_start_w));

		Dtype maxval = 0; 

		int maxidx = -1;
		bottom_data += (roi_batch_ind * channels + c) * height * width;
		for (int h = hstart; h < hend; ++h) {
			for (int w = wstart; w < wend; ++w) {
				if (!(w > roi_start_w_in && w < roi_end_w_in && h > roi_start_h_in && h < roi_end_h_in)) {
					// if it is not inside the inner rectangle of the region
					int bottom_index = h * width + w;
					if (bottom_data[bottom_index] > maxval) {
						maxval = bottom_data[bottom_index];
						maxidx = bottom_index;
					}
				}
			}
		}
		top_data[index] = maxval;
		argmax_data[index] = maxidx;
	}
}

template <typename Dtype>
__global__ void ROIPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* argmax_data, const int num_rois, const Dtype spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, Dtype* bottom_diff,
    const Dtype* bottom_rois) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		// (n, c, h, w) coords in bottom data
		int w = index % width;
		int h = (index / width) % height;
		int c = (index / width / height) % channels;
		int n = index / width / height / channels;

		Dtype gradient = 0;
		// Accumulate gradient over all ROIs that pooled this element
		for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
			const Dtype* offset_bottom_rois = bottom_rois + roi_n * 9;
			int roi_batch_ind = offset_bottom_rois[0];
			// Skip if ROI's batch index doesn't match n
			if (n != roi_batch_ind) {
				continue;
			}


			// outer rectangle of the region
			int roi_start_w   = int(offset_bottom_rois[1]);// * spatial_scale);
			int roi_start_h   = int(offset_bottom_rois[2]);// * spatial_scale);
			int roi_end_w     = int(offset_bottom_rois[3]);// * spatial_scale);
			int roi_end_h     = int(offset_bottom_rois[4]);// * spatial_scale);

			// inner rectangle of the region
			int roi_start_w_in= int(offset_bottom_rois[5]);// * spatial_scale);
			int roi_start_h_in= int(offset_bottom_rois[6]);// * spatial_scale);
			int roi_end_w_in  = int(offset_bottom_rois[7]);// * spatial_scale);
			int roi_end_h_in  = int(offset_bottom_rois[8]);// * spatial_scale);


			// Skip if ROI doesn't include (h, w)
			const bool in_roi =  (w >= roi_start_w && w <= roi_end_w &&
					h >= roi_start_h && h <= roi_end_h) && 
				!(w > roi_start_w_in && w < roi_end_w_in && 
						h > roi_start_h_in && h < roi_end_h_in);

			if (!in_roi) {
				continue;
			}

			int top_offset = (roi_n * channels + c) * pooled_height * pooled_width;
			const Dtype* offset_top_diff = top_diff + top_offset;
			const int* offset_argmax_data = argmax_data + top_offset;

			// Compute feasible set of pooled units that could have pooled
			// this bottom unit

			// Force malformed ROIs to be 1x1
			int roi_width = max(roi_end_w - roi_start_w + 1, 1);
			int roi_height = max(roi_end_h - roi_start_h + 1, 1);

			Dtype bin_size_h = static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
			Dtype bin_size_w = static_cast<Dtype>(roi_width)  / static_cast<Dtype>(pooled_width);

			int phstart = floor(static_cast<Dtype>(h - roi_start_h) / bin_size_h);
			int phend = ceil(static_cast<Dtype>(h - roi_start_h + 1) / bin_size_h);
			int pwstart = floor(static_cast<Dtype>(w - roi_start_w) / bin_size_w);
			int pwend = ceil(static_cast<Dtype>(w - roi_start_w + 1) / bin_size_w);

			phstart = min(max(phstart, 0), pooled_height);
			phend = min(max(phend, 0), pooled_height);
			pwstart = min(max(pwstart, 0), pooled_width);
			pwend = min(max(pwend, 0), pooled_width);

			for (int ph = phstart; ph < phend; ++ph) {
				for (int pw = pwstart; pw < pwend; ++pw) {
					if (offset_argmax_data[ph * pooled_width + pw] == (h * width + w)) {
						gradient += offset_top_diff[ph * pooled_width + pw];
					}
				}
			}
		}
		bottom_diff[index] = gradient;
	}
}

THCState* getCutorchState(lua_State* L)
{
    lua_getglobal(L, "cutorch");
    lua_getfield(L, -1, "getState");
    lua_call(L, 0, 1);
    THCState *state = (THCState*) lua_touserdata(L, -1);
    lua_pop(L, 2);
    return state;
}

static int updateOutput(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *rois = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
	THCudaIntTensor *argmax = (THCudaIntTensor *)luaT_getfieldcheckudata(L, 1, "argmax", "torch.CudaIntTensor");

	int pooled_height_ = luaT_getfieldcheckint(L, 1, "pooled_height");
	int pooled_width_ = luaT_getfieldcheckint(L, 1, "pooled_width");
	THCudaTensor_resize5d(state, output, THCudaTensor_size(state, rois, 0), THCudaTensor_size(state, rois, 1), THCudaTensor_size(state, input, 1), pooled_height_, pooled_width_);
	THCudaIntTensor_resize5d(state, argmax, THCudaTensor_size(state, rois, 0), THCudaTensor_size(state, rois, 1), THCudaTensor_size(state, input, 1), pooled_height_, pooled_width_);

	const float* bottom_data = THCudaTensor_data(state, input);
	const float* bottom_rois = THCudaTensor_data(state, rois);
	float* top_data = THCudaTensor_data(state, output);
	int* argmax_data = THCudaIntTensor_data(state, argmax); // int -> float
	
	// TODO: BATCH
	// BDHW 1DHW
	int count = THCudaTensor_nElement(state, output); // top[0]->count();
	int channels_ = THCudaTensor_size(state, input, 1);
	int height_ = THCudaTensor_size(state, input, 2);
	int width_ = THCudaTensor_size(state, input, 3);
	float spatial_scale_ = luaT_getfieldchecknumber(L, 1, "spatial_scale");

	CUDA_POST_KERNEL_CHECK;
	
	// NOLINT_NEXT_LINE(whitespace/operators)
	ROIPoolForward<float><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			  count, bottom_data, spatial_scale_, channels_, height_, width_, pooled_height_,
			  pooled_width_, bottom_rois, top_data, argmax_data);
	CUDA_POST_KERNEL_CHECK;

	return 1;
}

static int updateGradInput(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *rois = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaIntTensor *argmax = (THCudaIntTensor *)luaT_getfieldcheckudata(L, 1, "argmax", "torch.CudaIntTensor");

	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

	THCudaTensor_resizeAs(state, gradInput, input);
	THCudaTensor_zero(state, gradInput);
	
	const float* bottom_rois = THCudaTensor_data(state, rois);
	const float* top_diff = THCudaTensor_data(state, gradOutput);
	float* bottom_diff = THCudaTensor_data(state, gradInput);
	int* argmax_data = THCudaIntTensor_data(state, argmax);

	const int count = THCudaTensor_nElement(state, gradInput);
    int channels_ = THCudaTensor_size(state, input, 1);
	int height_ = THCudaTensor_size(state, input, 2);
	int width_ = THCudaTensor_size(state, input, 3);
	int pooled_height_ = luaT_getfieldcheckint(L, 1, "pooled_height");
	int pooled_width_ = luaT_getfieldcheckint(L, 1, "pooled_width");
	float spatial_scale_ = luaT_getfieldchecknumber(L, 1, "spatial_scale");
	int num_rois = THCudaTensor_size(state, rois, 0) * THCudaTensor_size(state, rois, 1); // bachSize x numRoisPerImage

	// NOLINT_NEXT_LINE(whitespace/operators)
	CUDA_POST_KERNEL_CHECK;
	ROIPoolBackward<float><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, top_diff, argmax_data, num_rois, spatial_scale_, channels_,
			height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
	CUDA_POST_KERNEL_CHECK;

	return 1;
}

static const struct luaL_Reg lua_registrations [] = {
  {"updateOutput", updateOutput},
  {"updateGradInput", updateGradInput},
  {NULL, NULL}
};

LUA_EXTERNC DLL_EXPORT int luaopen_libcucontextlocnet(lua_State *L)
{
  lua_newtable(L);

  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, lua_registrations, "contextlocnet");
  lua_pop(L,1);

  return 1;
}
