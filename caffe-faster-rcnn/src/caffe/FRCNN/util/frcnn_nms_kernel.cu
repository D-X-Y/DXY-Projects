#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include "caffe/FRCNN/util/frcnn_gpu_nms.hpp"
#include "caffe/common.hpp"
#include <vector>
#include <iostream>

namespace caffe {

namespace Frcnn {

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 3];
    //block_boxes[threadIdx.x * 5 + 4] =
    //  dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 4;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 4) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _set_device(int device_id) {
  if (device_id<=0) return;
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) {
    return;
  }
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  CUDA_CHECK(hipSetDevice(device_id));
}

void gpu_nms(int* keep_out, int* num_out, const float* boxes_dev, int boxes_num,
          int boxes_dim, float nms_overlap_thresh, int device_id) {
  _set_device(device_id);

  // float* boxes_dev = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  // CUDA_CHECK(hipMalloc(&boxes_dev,
  //                       boxes_num * boxes_dim * sizeof(float)));
  // CUDA_CHECK(hipMemcpy(boxes_dev,
  //                       boxes_host,
  //                       boxes_num * boxes_dim * sizeof(float),
  //                       hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;
    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  // CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(mask_dev));
}

} // namespace frcnn

} // namespace caffe
