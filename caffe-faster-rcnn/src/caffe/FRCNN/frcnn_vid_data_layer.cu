#include "caffe/FRCNN/frcnn_vid_data_layer.hpp"

namespace caffe {

namespace Frcnn {

template <typename Dtype>
void FrcnnVidDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = this->prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data, Image Blob
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(), top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // top[1] is image_info , top[2] is gx_bpxes
    caffe_copy(3, batch->label_.gpu_data(), top[1]->mutable_gpu_data());
    // Reshape to loaded labels.
    top[2]->Reshape(batch->label_.num()-1, batch->label_.channels(), batch->label_.height(), batch->label_.width());
    // Copy the labels.
    // First five is image_info
    caffe_copy(batch->label_.count() - 5, batch->label_.gpu_data() + 5, top[2]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  this->prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(FrcnnVidDataLayer);

}  // namespace Frcnn

}  // namespace caffe
